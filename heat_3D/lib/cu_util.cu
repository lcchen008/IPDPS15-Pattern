#include "cu_util.h"

int GetGPUNumber()
{
        int count;

        hipGetDeviceCount(&count);
	//    printf("Number of devices is: %d\n", count);

        //if(count==0)
        //{
        //        fprintf(stderr, "There is no device.\n");
        //        return 0;
        //}

	//int counter = 0;

        //int i;
        //for(i = 0; i<count; i++)
        //{
        //        hipDeviceProp_t prop;
        //        if(hipGetDeviceProperties(&prop, i)==hipSuccess)
        //        {
        //                if(prop.major>=2)
        //                {
        //                    //check whether support zero-copy
        //                    bool whe = prop.unifiedAddressing;

        //                    if(whe)
        //                    {
	//			//printf("%d supports unified addressing\n");
	//			counter++;
        //                    }

        //                    //break;
        //                }
        //        }
	//}       

    	//hipSetDevice(0);
	//return 1;
     	return count;      
}

void checkCUDAError(const char *msg)
{
        hipError_t err = hipGetLastError();
        if( hipSuccess != err)
        {
                fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
                exit(EXIT_FAILURE);
        }
}

