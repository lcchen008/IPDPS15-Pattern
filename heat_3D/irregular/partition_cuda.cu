#include "partition_cuda.h"
#include "../lib/macro.h"
#include <stdio.h> 
#include <stdlib.h> 

void partition_cuda::Allocate()
{
	//node data come from zero-copy	

	//allocate edges
	hipMalloc((void **)&my_edges_d_, sizeof(EDGE)*my_num_edges_);
	//copy in edge_data
	hipMemcpy(my_edges_d_, my_edges_, sizeof(EDGE)*my_num_edges_, hipMemcpyHostToDevice);

	CUDA_SAFE_CALL(hipMalloc((void **)&my_node_data_device_, my_num_nodes_ * node_data_elm_size_));				
	CUDA_SAFE_CALL(hipMemcpy(my_node_data_device_, my_node_data_d_, my_num_nodes_ * node_data_elm_size_, hipMemcpyHostToDevice));

	//allocate edge data
	my_edge_data_ = NULL;
	my_edge_data_d_ = NULL;

	if(edge_data_elm_size_!=0)
	{
		hipMalloc((void **)&my_edge_data_d_, edge_data_elm_size_*my_num_edges_);

		hipMemcpy(my_edge_data_d_, my_edge_data_, edge_data_elm_size_*my_num_edges_, hipMemcpyHostToDevice);
	}

	//allocate parts
	hipMalloc((void **)&my_parts_d_, sizeof(Part) * my_num_parts_);
	//copy in parts
	hipMemcpy(my_parts_d_, my_parts_, sizeof(Part) * my_num_parts_, hipMemcpyHostToDevice);
}

partition_cuda::~partition_cuda()
{
	//hipHostFree(my_node_data_d_);	

	if(my_edge_data_d_)
		hipFree(my_edge_data_d_);

	hipFree(my_edges_d_);
	free(my_parts_);
	hipFree(my_parts_d_);
}
